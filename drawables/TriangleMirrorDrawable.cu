#include "TriangleMirrorDrawable.cuh"


__device__ TriangleMirrorDrawable::TriangleMirrorDrawable(Vector3D point1, Vector3D point2, Vector3D point3, Color6Component c, int subType) :
	A(point1), B(point2), C(point3), color(c), type(subType) // constructeur - d�finition
{
	AB = B.sub(A);
	AC = C.sub(A);
	normal = AB.cross(AC);
}

__device__ TriangleMirrorDrawable::TriangleMirrorDrawable() :
	A(Vector3D()), B(Vector3D()), C(Vector3D()), color(Color6Component()), type(0)
{
	normal = Vector3D(0, 1, 0);
	AB = Vector3D();
	AC = Vector3D();
}

__device__ TriangleMirrorDrawable::TriangleMirrorDrawable(const Drawable& obj) : Drawable(obj), type(0)
{

}

__device__ TriangleMirrorDrawable::~TriangleMirrorDrawable() // Destructeur - d�finition
{

}

__device__ HitRecord TriangleMirrorDrawable::hit(Ray r, float tmin, float tmax)
{
	HitRecord rec = HitRecord();
	float det = -r.getDirection().dot(normal);
	float invdet = 1.0 / det;
	Vector3D AO = r.getOrigin().sub(A);
	Vector3D DAO = AO.cross(r.getDirection());
	rec.u = AC.dot(DAO)*invdet;
	rec.v = -AB.dot(DAO)*invdet;
	float t = AO.dot(normal)*invdet;
	if (det >= 1e-6 && t >= tmin && t <= tmax && rec.u >= 0.0 && rec.v >= 0.0 && rec.u + rec.v <= 1.0) {
		rec.normal = r.getDirection().sub(normal.unitVector().mul((r.getDirection().dot(normal.unitVector())*2.0)));
		rec.point = r.getOrigin().add(r.getDirection().mul(t));
		rec.t = t;
		rec.front_face = true;
		rec.isEmpty = false;
	}
	return rec;
}

__device__ Color6Component TriangleMirrorDrawable::getColor(HitRecord* hit)
{
	return color;
}

__device__ bool TriangleMirrorDrawable::doReflect()
{
	return true;
}

__device__ int TriangleMirrorDrawable::getSubType() {
	return type;
}